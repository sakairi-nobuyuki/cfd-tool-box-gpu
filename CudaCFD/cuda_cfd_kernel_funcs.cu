#include "hip/hip_runtime.h"
#ifndef __CUDA_FRONT_END__
#define __CUDA_FRONT_END__

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <assert.h>

#include "cuda_cfd_kernel_funcs.h"

using namespace std;


__host__ void allocate_cuda_memory(void **U, int n_bytes) {
//    printf("  In cuda_cfd_kernel_functions, allocate_cuda_memory, before allocating memory address of U: %p\n", &U);
    //printf("  In cuda_cfd_kernel_functions, allocate_cuda_memory, before allocating memory address of V:  %p\n", V);
    hipMalloc (U, n_bytes);
//    printf("  In cuda_cfd_kernel_functions, allocate_cuda_memory, after allocating memory address of U:  %p\n", &U);
}

__host__ void copy_memory_host_to_device(double *gU, double *U, int n_bytes) {
    hipMemcpy (gU, U, n_bytes, hipMemcpyHostToDevice);
}

__host__ void copy_memory_device_to_host(double *U, double *gU, int n_bytes) {
    
    hipMemcpy (U, gU, n_bytes, hipMemcpyDeviceToHost);

}

__host__ void copy_memory_device_to_device(double *gV, double *gU, int n_bytes) {

    hipMemcpy (gV, gU, n_bytes, hipMemcpyDeviceToDevice);
}

__host__ void copy_memory_mock() {
    double *gU, *U, *V;
    int i, n;

    printf("In memory test mock, before memory allocation, address U: %p, V: %p, gU: %p\n", U, V, gU);

    n = 10;
    U = (double *) malloc(sizeof(double) * n);
    V = (double *) malloc(sizeof(double) * n);
    hipMalloc((double **) &gU, sizeof(double) * n);

    printf("In memory test mock, after memory allocation, address U: %p, V: %p, gU: %p\n", U, V, gU);

    for (i = 0; i < n; i++) U[i] = (double) i;
    printf("In mock test, U\n  ");
    for (i = 0; i < n; i++) printf("%lf  ", U[i]);
    printf("\n");

    hipMemcpy(gU, U, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(V, gU, sizeof(double) * n, hipMemcpyDeviceToHost);
    printf("In mock test, V\n  ");
    for (i = 0; i < n; i++) printf("%lf  ", V[i]);
    printf("\n");
    printf("In mock test, assert U == V\n  ");
    for (i = 0; i < n; i++) assert(U[i] == V[i]);
    printf("\n");

    printf("In memory test mock, after test memory allocation, address U: %p, V: %p, gU: %p\n", U, V, gU);
}

void cuda_device_synchronize() {
    hipDeviceSynchronize();
}


__global__ void obtain_delta_plus_device(double *gU, double *gDelta, int n_len) {
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (0 < i && i < n_len) gDelta[i] = gU[i + 1] - gU[i];
    if (i == 0)             gDelta[i] = gU[i + 1] - gU[i];
    if (i == n_len - 1)     gDelta[i] = gDelta[i - 1];
}

__global__ void obtain_delta_minus_device(double *gU, double *gDelta, int n_len) {
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (0 < i && i < n_len) gDelta[i] = gU[i] - gU[i - 1];
    if (i == 0)             gDelta[i] = gDelta[i + 1];
    if (i == n_len - 1)     gDelta[i] = gU[i] - gU[i - 1];
}

//void obtain_deltas_device(double *gU, double *gDeltaPlus, double *gDeltaMinus, int n_len) {
void obtain_deltas_device(double *gU, double *gDeltaPlus, double *gDeltaMinus, GridDim *dimGrid, BlockDim *dimBlock, int n_len) {
    dim3 grid(dimGrid->x, dimGrid->y), block(dimBlock->x, dimBlock->y, dimBlock->z);
    obtain_delta_plus_device<<<grid, block>>>(gU, gDeltaPlus, n_len);
    obtain_delta_minus_device<<<grid, block>>>(gU, gDeltaMinus, n_len);

}



__host__ void free_cuda_memory(double *U) {
    hipFree(U);
}



#endif