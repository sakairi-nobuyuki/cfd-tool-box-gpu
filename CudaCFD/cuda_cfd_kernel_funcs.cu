#include "hip/hip_runtime.h"
#ifndef __CUDA_FRONT_END__
#define __CUDA_FRONT_END__

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <assert.h>

#include "cuda_cfd_kernel_funcs.h"

using namespace std;


__host__ void allocate_cuda_memory(void **U, int n_bytes) {
//    printf("  In cuda_cfd_kernel_functions, allocate_cuda_memory, before allocating memory address of U: %p\n", &U);
    //printf("  In cuda_cfd_kernel_functions, allocate_cuda_memory, before allocating memory address of V:  %p\n", V);
    hipMalloc (U, n_bytes);
//    printf("  In cuda_cfd_kernel_functions, allocate_cuda_memory, after allocating memory address of U:  %p\n", &U);
}

__host__ void copy_memory_host_to_device(double *gU, double *U, int n_bytes) {
    hipMemcpy (gU, U, n_bytes, hipMemcpyHostToDevice);
}

__host__ void copy_memory_device_to_host(double *U, double *gU, int n_bytes) {
    
    hipMemcpy (U, gU, n_bytes, hipMemcpyDeviceToHost);

}

__host__ void copy_memory_device_to_device(double *gV, double *gU, int n_bytes) {

    hipMemcpy (gV, gU, n_bytes, hipMemcpyDeviceToDevice);
}



void cuda_device_synchronize() {
    hipDeviceSynchronize();
}


__global__ void obtain_delta_plus_device(double *gDelta, double *gU, int n_len) {
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (0 < i && i < n_len) gDelta[i] = gU[i + 1] - gU[i];
    if (i == 0)             gDelta[i] = gU[i + 1] - gU[i];
    if (i == n_len - 1)     gDelta[i] = gDelta[i - 1];
}

__global__ void obtain_delta_minus_device(double *gDelta, double *gU, int n_len) {
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (0 < i && i < n_len) gDelta[i] = gU[i] - gU[i - 1];
    if (i == 0)             gDelta[i] = gDelta[i + 1];
    if (i == n_len - 1)     gDelta[i] = gU[i] - gU[i - 1];
}

//void obtain_deltas_device(double *gU, double *gDeltaPlus, double *gDeltaMinus, int n_len) {
void obtain_deltas_device(double *gDeltaPlus, double *gDeltaMinus, double *gU, GridDim *dimGrid, BlockDim *dimBlock, int n_len) {
    dim3 grid(dimGrid->x, dimGrid->y), block(dimBlock->x, dimBlock->y, dimBlock->z);
    obtain_delta_plus_device<<<grid, block>>>(gDeltaPlus, gU, n_len);
    obtain_delta_minus_device<<<grid, block>>>(gDeltaMinus, gU, n_len);

}


__global__ void obtain_minmod(double *gBarDeltaPlus, double *gBarDeltaMinus, double *gDeltaPlus, double *gDeltaMinus, double b, int n_len) {
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (0 <= i && i < n_len) 
        gBarDeltaPlus[i] 
            = copysignf(1.0, gDeltaPlus[i]) 
            * fmaxf(0.0, fminf(fabs(gDeltaPlus[i]), copysignf(1.0, gDeltaPlus[i]) * b * gDeltaMinus[i]));
    if (0 <= i && i < n_len) 
        gBarDeltaMinus[i] 
            = copysignf(1.0, gDeltaMinus[i]) 
            * fmaxf(0.0, fminf(fabs(gDeltaMinus[i]), copysignf(1.0, gDeltaMinus[i]) * b * gDeltaPlus[i]));

}

void obtain_minmod_device(double *gBarDeltaPlus, double *gBarDeltaMinus, double *gDeltaPlus, double *gDeltaMinus, double b, GridDim *dimGrid, BlockDim *dimBlock, int n_len) {
    dim3 grid(dimGrid->x, dimGrid->y), block(dimBlock->x, dimBlock->y, dimBlock->z);
    obtain_minmod<<<grid, block>>>(gBarDeltaPlus, gBarDeltaMinus, gDeltaPlus, gDeltaMinus, b, n_len);

}


__global__ void obtain_slope(double *Slope, double *DeltaPlus, double *DeltaMinus, double epsilon, int n_len) {
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (0 <= i && i < n_len) Slope[i] = (2.0 * DeltaPlus[i] * DeltaMinus[i] + epsilon) 
        / (pow(DeltaPlus[i], 2.0) + pow(DeltaMinus[i], 2.0) + epsilon);
}

void obtain_slope_device(double *Slope, double *DeltaPlus, double *DeltaMinus, double epsilon, GridDim *dimGrid, BlockDim *dimBlock, int n_len) {
    dim3 grid(dimGrid->x, dimGrid->y), block(dimBlock->x, dimBlock->y, dimBlock->z);
    obtain_slope<<<grid, block>>>(Slope, DeltaPlus, DeltaMinus, epsilon, n_len);
}


__global__ void obtain_cell_intface_values(double *R, double *L, double *Q, double *DeltaPlus, double *DeltaMinus, double *s, double kappa, int n_len) {
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (0 <= i && i < n_len - 1) L[i] = Q[i] + 0.25 * s[i] * ((1.0 - kappa * s[i]) * DeltaMinus[i] + (1.0 + kappa) * s[i] * DeltaPlus[i]);
    if (i == n_len - 1) L[n_len - 1] = L[n_len - 2];

    if (0 < i && i < n_len) R[i] = Q[i+1] - 0.25 * s[i+1] * ((1.0 - kappa * s[i+1]) * DeltaPlus[i+1] + (1.0 + kappa * s[i+1]) * DeltaMinus[i+1]);
    if (i == 0) R[0] = R[1];
}

void obtain_cell_intface_value_device(double *R, double *L, double *Q, double *DeltaPlus, double *DeltaMinus, double *s, double kappa, GridDim *dimGrid, BlockDim *dimBlock, int n_len) {
    dim3 grid(dimGrid->x, dimGrid->y), block(dimBlock->x, dimBlock->y, dimBlock->z);
    obtain_cell_intface_values<<<grid, block>>>(R, L, Q, DeltaPlus, DeltaMinus, s, kappa, n_len);
}

void obtain_cell_intface_value_from_Q_device(double *R, double *L, double *s, double *BarDeltaPlus, double *BarDeltaMinus, double *DeltaPlus, double *DeltaMinus, double *Q, double kappa, double epsilon, double b, GridDim *dimGrid, BlockDim *dimBlock, int n_len) {
    dim3 grid(dimGrid->x, dimGrid->y), block(dimBlock->x, dimBlock->y, dimBlock->z);
    hipDeviceSynchronize();
    obtain_delta_plus_device<<<grid, block>>>(DeltaPlus, Q, n_len);
    obtain_delta_minus_device<<<grid, block>>>(DeltaMinus, Q, n_len);
    hipDeviceSynchronize();
    obtain_minmod<<<grid, block>>>(BarDeltaPlus, BarDeltaMinus, DeltaPlus, DeltaMinus, b, n_len);
    hipDeviceSynchronize();
    obtain_slope<<<grid, block>>>(s, BarDeltaPlus, BarDeltaMinus, epsilon, n_len);
    hipDeviceSynchronize();
    obtain_cell_intface_values<<<grid, block>>>(R, L, Q, DeltaPlus, DeltaMinus, s, kappa, n_len);    
    hipDeviceSynchronize();
}

__host__ void free_cuda_memory(double *U) {
    hipFree(U);
}



#endif