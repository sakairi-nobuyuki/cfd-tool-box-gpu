
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NN 20

void output_result (double *U, int n_array, int n_iter) {
    int i;
    FILE *fp_out;
    char file_name[64];

    sprintf (file_name, "%05d.dat", n_iter);
    printf ("output result to: %s\n", file_name);

    if ((fp_out = fopen (file_name, "w")) == NULL) {
        printf ("cannot open %s\n", file_name);
        exit (1);
    }

    for (i = 0; i < n_array; i++) fprintf (fp_out, "%d\t%lf\n", i, U[i]);

    fclose (fp_out);

}


__global__ void sum_array (double *array_1, double *array_2, double *array_3, int n_array) {
    int i, j, n;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    n = 10;

    for (j = 0; j < n; j++) {
        //if (i < NN)  array_3[i] = array_1[i] + array_3[i];        
        array_3[i] = 2.0 * array_3[i];
    }
    
}


__global__ void ones (double *U) {
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    U[i] = i;
}


__global__ void derivertive_array (double *array_in, double *array_out, int n_array) {
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (0 < i && i < NN - 1) array_out[i] = array_in[i] - array_in[i-1];
    if (i == 0) array_out[i] = array_in[i + 1];
    if (i == NN - 1) array_out[i] = array_in[i - 1];
}

__global__ void solve_diffusion_eq (double *array_in, double *array_out, double rdx2, double dt, int n_array) {
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (0 < i && i < NN - 1) array_out[i] = array_out[i] = array_in[i] + 0.5 * (array_in[i + 1] - 2.0 * array_in[i] + array_in[i - 1]) * rdx2 * dt;
    if (i == 0) array_out[i] = array_in[i + 1];
    if (i == NN - 1) array_out[i] = array_in[i - 1];
}

__global__ void renew_vers (double *array_in, double *array_out) {
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    array_out[i] = array_in[i];
}


void initialize_array (double *array, int size) {
    int i;

//    for (i = 0; i < NN; i++)  array[i] = (double) rand ();
    //for (i = 0; i < NN; i++)  array[i] = 1.0;
    //for (i = 0; i < NN; i++)  array[i] = (double) i;

    for (i = 0; i < NN; i++) {
        if (i < NN / 3 || 2 * NN/ 3 < i) array[i] = 0.0;
        else array[i] = 1.0;
    }

}

void print_result (double *array, int n) {
    int i;


    //for (i = 0; i < n; i++)  printf ("%.0lf ", array[i]);
    for (i = 0; i < n; i++)  printf ("%.3lf ", array[i]);
    printf ("\n");

}

int main () {
    int i, n;
    double *H, *Htmp, *HU, *HUtmp;   // SWE concerning vars in CPU
    double *U, *Utmp;                // supplementary vars in CPU
    double *gU, *gUtmp;              // SWE concerning vars in GPU
    size_t n_bytes = NN * sizeof (double);   // size of memory allocation for calculation area
    time_t start_time, end_time;
    dim3 Grid, Block;               //  number of grid and block in CUDA

    Grid.x = NN / 196 + 1;
    Block.x = 196;

    printf ("start calc\n");
    start_time = time (NULL);

    H    = (double *) malloc (n_bytes);
    Htmp = (double *) malloc (n_bytes);
    HU    = (double *) malloc (n_bytes);
    HUtmp = (double *) malloc (n_bytes);
    U    = (double *) malloc (n_bytes);
    Utmp = (double *) malloc (n_bytes);

    printf ("memory allocation finished\n");

    initialize_array (U, n_bytes);
    initialize_array (Utmp, n_bytes);

    printf ("initialize memory\n");
    printf ("cuda memory allocation\n");

    hipMalloc ((void**) &gU,    n_bytes);
    hipMalloc ((void**) &gUtmp, n_bytes);

    printf ("cuda memory allocation finished\n");
    printf ("cuda memory copy\n");

    hipMemcpy (gU,    U,    n_bytes, hipMemcpyHostToDevice);
    hipMemcpy (gUtmp, Utmp, n_bytes, hipMemcpyHostToDevice);

    printf ("cuda memory copy finished\n");

    printf ("inp array1\n");
    print_result (U, NN);

    printf ("start kernel function\n");
    //sum_array<<<Grid, Block>>> (d_array_1, d_array_2, d_array_3, n_bytes);
    derivertive_array<<<Grid, Block>>> (gU, gUtmp, n_bytes);
    hipMemcpy (U, gU, n_bytes, hipMemcpyDeviceToHost);
    //cudaMemcpy (Utmp, gUtmp, n_bytes, cudaMemcpyDeviceToHost);
    printf ("derivertice test\n");
    //print_result (Utmp, NN);
    print_result (U, NN);

    for (n = 0; n < 100000; n++) {
        if (n % 1000 == 0) {
            hipMemcpy (U, gU, n_bytes, hipMemcpyDeviceToHost);
            print_result (U, NN);
            output_result (U, NN, n);
        }
        solve_diffusion_eq <<<Grid, Block>>> (gU, gUtmp, 1.0, 0.1, n_bytes);
        hipDeviceSynchronize();
        renew_vers <<<Grid, Block>>> (gUtmp, gU);
        hipDeviceSynchronize();
     }

    hipDeviceSynchronize();
    printf ("end kernel function\n");
    hipMemcpy (U, gU, n_bytes, hipMemcpyDeviceToHost);
    printf ("res array3\n");
    print_result (U, NN);

    end_time = time (NULL);
    
    printf ("calc time: %ld\n", end_time - start_time);

    return 0;
}